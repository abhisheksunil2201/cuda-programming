
#include <hip/hip_runtime.h>
#include<iostream>

__global__ void add(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

//host = CPU and device = GPU
int main() {
    int N = 4096;
    int BLOCK_SIZE = 256;
    float *a = new float[N];
    float *b = new float[N];
    float *c = new float[N];
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2*i;
    }
    float *a_d;
    float *b_d;
    float *c_d;

    //allocating device memory
    hipMalloc((void**) &a_d, N*sizeof(float));
    hipMalloc((void**) &b_d, N*sizeof(float));
    hipMalloc((void**) &c_d, N*sizeof(float));

    //copy data from host to device
    hipMemcpy(a_d, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, N*sizeof(float), hipMemcpyHostToDevice);

    //launching kernel
    add<<<ceil(N/(float)BLOCK_SIZE), BLOCK_SIZE>>>(a_d, b_d, c_d, N);

    //copy data from device to host
    hipMemcpy(c, c_d, N*sizeof(float), hipMemcpyDeviceToHost);

    //printing result
    for (int i = 0; i < 10; i++) {
        std::cout<<a[i]<<" "<<b[i]<<" "<<c[i]<<std::endl;
    }  

    //freeing device memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}